#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

			//thrust::host_vector<int> host_thrust_in = idata;
			//thrust::host_vector<int> host_thrust_out = odata;

			//thrust::host_vector<int> host_thrust_in(n);
			
			//thrust::device_vector<int> dev_thrust_in = host_thrust_in;
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

			thrust::exclusive_scan(idata, idata + n, odata);
            timer().endGpuTimer();
        }
    }
}
