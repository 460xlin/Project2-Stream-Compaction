#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>
namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


		__global__ void kernelScanReduce(int n, int d, int* odata, int* idata)
		{
			int thID = threadIdx.x + blockDim.x * blockIdx.x;
			if (thID >= n) return;
			int temp = 1 << d;
			int temp2 = 1 << (d - 1);
			
			if ((thID % temp) == 0)
			{
				odata[thID + temp - 1] = idata[thID + temp2 - 1] + idata[thID + temp - 1];
			}
			else
			{
				odata[thID] = idata[thID];
			}
		}

		__global__ void kernelScanDownSweep(int n, int d, int* odata, int* idata)
		{
			int thID = threadIdx.x + blockDim.x * blockIdx.x;
			if (thID >= n) return;
			int tempdp1 = 1 << (d + 1);
			int tempd = 1 << d;
			if ((thID % tempdp1) == 0)
			{
				int t = idata[thID + tempd - 1];
				odata[thID + tempd - 1] = idata[thID + tempdp1 - 1];
				odata[thID + tempdp1 - 1] = t + idata[thID + tempdp1 - 1];

			}
			else
			{
				odata[thID] = idata[thID];
			}
		}
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
			int temp = 1 << ilog2ceil(n);
			std::cout << "my temp number is " << temp << std::endl;
			std::cout << "my ilog2ceil(n) is " << ilog2ceil(n) << std::endl;
			std::cout << "my n is " << n << std::endl;
			int myIdentity = 0;

			dim3 fullBlocksPerGrid((temp + blockSize - 1) / blockSize);
			int* dev_In = NULL;
			int* dev_Out = NULL;

			hipMalloc((void**)&dev_In, temp * sizeof(int));
			checkCUDAError("Malloc dev_In failed!");
			hipMalloc((void**)&dev_Out, temp * sizeof(int));
			checkCUDAError("Malloc dev_Out failed!");
			hipMemcpy(dev_In, idata, temp * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("Memcpy from idata to dev_In failed!");

			std::cout << "still works before kernel scan reduce" << std::endl;
			for (int d = 1; d <= ilog2ceil(n); ++d)
			{
				std::cout << "d is " << d << std::endl;
				kernelScanReduce << <fullBlocksPerGrid, blockSize >> > (temp, d, dev_Out, dev_In);
				std::swap(dev_Out, dev_In);

			}
			std::swap(dev_Out, dev_In);
			dev_Out[n - 1] = myIdentity;
			//
			//for (int d = ilog2ceil(n) - 1; d >= 0; --d)
			//{
			//	kernelScanDownSweep << <fullBlocksPerGrid, blockSize >> > (temp, d, dev_Out, dev_In);
			//	std::swap(dev_Out, dev_In);
			//}
			//
			//std::swap(dev_Out, dev_In);
			hipMemcpy(odata, dev_Out, temp * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("Memcoy from dev_Out to odata failed!");

			for (int i = 0; i < temp; ++i)
			{
				std::cout << odata[i] << " ";
			}
            timer().endGpuTimer();

			hipFree(dev_In);
			hipFree(dev_Out);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
